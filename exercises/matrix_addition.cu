
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void addition(float* d_A, float* d_B, float* d_C, int numElements){
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	float tmp_1, tmp_2;
	if(row < numElements && col < numElements) {
		tmp_1 = d_A[col];
		__syncthreads();
		tmp_2 = d_B[col];
		__syncthreads();
		d_C[row] = tmp_1 + tmp_2;
		__syncthreads();
	}
}

__global__
void additionByRows(float* d_A, float* d_B, float* d_C, int numRows) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
	float tmp_1, tmp_2;
	if(row<numRows && col<numRows){
		for(int i=0; i<numRows; i++){
			tmp_1 = d_A[row+i*col];
                	__syncthreads();
                	tmp_2 = d_B[row+i*col];
                	__syncthreads();
               		d_C[row] = tmp_1 + tmp_2;
                	__syncthreads();		
		}
	}
}

__global__
void additionByColumns(float* d_A, float* d_B, float* d_C, int numColumns) {
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;
        float tmp_1, tmp_2;
        if(row<numColumns && col<numColumns){
                for(int i=0; i<numColumns; i++){
                        tmp_1 = d_A[row*i+col];
                        __syncthreads();
                        tmp_2 = d_B[row*i+col];
                        __syncthreads();
                        d_C[row] = tmp_1 + tmp_2;
                        __syncthreads();
                }
        }
}

int main() {

	// size of matrixes
	int numArows = 512;
	int numAcolumns = 512;
	int numBcolumns = 512;

	float *h_A;
	float *h_B;
	float *h_C;

	float *d_A;
	float *d_B;
	float *d_C;

	// allocate data in host
	h_A = (float*) malloc( numArows*numAcolumns*sizeof(float));
	h_B = (float*) malloc( numAcolumns*numBcolumns*sizeof(float));
	h_C = (float*) malloc( numArows*numBcolumns*sizeof(float));

	// matrix initialization
	for(int i=0; i<numArows*numAcolumns; i++)
		h_A[i] = i;
	for(int i=0; i<numAcolumns*numBcolumns; i++)
		h_B[i] = i;

	// allocate data in device
	hipMalloc((void**)&d_A, numArows*numAcolumns*sizeof(float));
	hipMalloc((void**)&d_B, numAcolumns*numBcolumns*sizeof(float));
	hipMalloc((void**)&d_C, numArows*numBcolumns*sizeof(float));
	
	// copy inputs to device
	hipMemcpy(d_A, h_A, numArows*numAcolumns*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B, numAcolumns*numBcolumns*sizeof(float),hipMemcpyHostToDevice);
	
	// launch kernel
	int threadsPerBlock = 32;
	int n_blocks = ceil(numAcolumns*numAcolumns/32.0);
	
	addition<<<n_blocks, threadsPerBlock>>>(d_A,d_B,d_C, numAcolumns*numAcolumns);
	
	// copy output to host
	hipMemcpy(h_C, d_C, numArows*numBcolumns*sizeof(float),hipMemcpyDeviceToHost);

	// freeing space
	free(h_A);
	free(h_B);
	free(h_C);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
}
