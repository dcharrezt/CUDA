#include <stdio.h>
#include <hip/hip_runtime.h>
#include <math.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

int main(int argc, char** argv) {

    // Print the vector length to be used, and compute its size
    int numElements = strtol(argv[1], NULL, 10);
    size_t size = numElements * sizeof(float);
    printf("[Vector addition of %d elements]\n", numElements);

    // Variables to do the timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Allocate the host vectors
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device vectors
    float *d_A = NULL;
    hipMalloc((void **)&d_A, size);

    float *d_B = NULL;
    hipMalloc((void **)&d_B, size);

    float *d_C = NULL;
    hipMalloc((void **)&d_C, size);

    // Copy the host vectors to the device vectors
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int n_blocks = ceil(numElements/256.0);
    printf("CUDA kernel launch with %d blocks of %d threads\n", 
        n_blocks, threadsPerBlock);

    printf("Inactive Threads %d \n", (n_blocks*threadsPerBlock)-numElements);

    hipEventRecord(start);
    vectorAdd<<< n_blocks, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    hipEventRecord(stop);

    // Copy the device result vector to host result vector
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    // Blocks CPU execution until stop has been recorded
    hipEventSynchronize(stop);

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
            printf("Failed at", i);
        }
    }

    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    printf("Elapsed Time: %f milliseconds\n", milliseconds);

    // Destroying events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device global memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}
